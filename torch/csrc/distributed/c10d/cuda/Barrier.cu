#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/csrc/distributed/c10d/symm_mem/CUDASymmetricMemory-inl.h>
#include <torch/csrc/distributed/c10d/cuda/Barrier.cuh>

namespace c10d::cuda::detail {

__global__ void kernel_barrier(int32_t* value, size_t timeout_ms) {
  size_t start = c10d::symmetric_memory::global_timer_ns();
  size_t timeout_ns = timeout_ms * 1e6; // Convert milliseconds to nanoseconds
  while (true) {
    // Atomically read the value
    int current_value = atomicAdd(&value[0], 0);
    // Check if the value is equal to the expected value
    if (current_value == 1) {
      value[1] = 1;
      return;
    }

    if (timeout_ms > 0) {
      // Check if timeout has been reached
      size_t now = c10d::symmetric_memory::global_timer_ns();
      if ((now - start) > timeout_ns) {
        value[1] = 2;
        return;
      }
    }

    // sleep for 1ms
    __nanosleep(1000000);
  }
}

void BarrierHandle::run() {
  kernel_barrier<<<1, 1>>>(comm_.mutable_data_ptr<int32_t>(), timeout_.count());
}

BarrierHandle barrier(std::chrono::milliseconds timeout) {
  BarrierHandle handle{timeout};
  return handle;
}

} // namespace c10d::cuda::detail
